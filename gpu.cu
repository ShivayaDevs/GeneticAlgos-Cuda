#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <time.h>
using namespace std;

#define THREADS_PER_BLOCK 100
#define NUM_BLOCKS 1

typedef double HighlyPrecise;

const int GENOME_LENGTH = 14;

const float MUTATION_FACTOR = 0.2;
const float CROSSOVER_RATE = 0.6;

const int NUM_EPOCHS = 1000;

struct Chromosome {
	HighlyPrecise genes[GENOME_LENGTH];
	HighlyPrecise fitnessValue;
};

__global__ void setupRandomStream(unsigned int seed, hiprandState* states) {
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(seed, threadIndex, 0, &states[threadIndex]);
}

__device__ HighlyPrecise getFitnessValue(HighlyPrecise chromosome[]) {
	HighlyPrecise fitnessValue = 0;
	for (int i = 0; i < GENOME_LENGTH; i++) {
		fitnessValue += chromosome[i] * chromosome[i];
	}
	return fitnessValue;
}

/**
 * Sorts the population that is present in the shared memory of one block.
 * Please note that this sorting is sequential.
 */
__device__ void bubbleSort(Chromosome a[]) {
	int n = THREADS_PER_BLOCK;
	Chromosome temp;
	for (int i = 0; i < n; i++) {
		bool changed = false;
		for (int j = 0; j < n - 1 - i; j++) {
			if (a[j].fitnessValue > a[j + 1].fitnessValue) {
				temp = a[j + 1];
				a[j + 1] = a[j];
				a[j] = temp;
				changed = true;
			}
		}
		if (!changed) {
			break;
		}
	}
}

__device__ void printChromosome(Chromosome c) {
	printf("Fitness: %lf | Chromosome: ", c.fitnessValue);
	for (int j = 0; j < GENOME_LENGTH; j++) {
		printf("%lf ,", c.genes[j]);
	}
	printf("\n");
}

/**
 * Prints the whole population of a block from the shared memory.
 */
__device__ void printBlockPopulation(Chromosome blockPopulation[]) {
	for (int i = 0; i < THREADS_PER_BLOCK; i++) {
		printChromosome(blockPopulation[i]);
	}
}

__device__ void initializeBlockPopulation(Chromosome blockPopulation[],
		hiprandState* randomState) {
	HighlyPrecise chromosome[GENOME_LENGTH];
	for (int i = 0; i < GENOME_LENGTH; i++) {
		chromosome[i] = 2.0 * hiprand_uniform(randomState) - 1;
		blockPopulation[threadIdx.x].genes[i] = chromosome[i];
	}
	blockPopulation[threadIdx.x].fitnessValue = getFitnessValue(chromosome);
}

__device__ Chromosome crossover(Chromosome blockPopulation[],
		hiprandState* randomState, int num_parents) {
	// Choosing parents.
	int maleIndex = hiprand_uniform(randomState) * num_parents;
	int femaleIndex = hiprand_uniform(randomState) * num_parents;
	if (maleIndex == femaleIndex) {
		return blockPopulation[threadIdx.x];
	}
	Chromosome male = blockPopulation[maleIndex];
	Chromosome female = blockPopulation[femaleIndex];
	Chromosome offspring;

	for (int i = 0; i < GENOME_LENGTH; i++) {
		offspring.genes[i] =
				(i < GENOME_LENGTH / 2) ? male.genes[i] : female.genes[i];
	}
	return offspring;
}

__device__ void mutate(Chromosome *offspring, hiprandState* randomState) {
	for (int i = 0; i < GENOME_LENGTH; i++) {
		HighlyPrecise multiplier = (2.0 * hiprand_uniform(randomState) - 1) / 10;
		offspring->genes[i] *= multiplier;
	}
}

__device__ void startIteration(Chromosome blockPopulation[],
		hiprandState* randomState) {

	int num_parents = THREADS_PER_BLOCK * (1 - CROSSOVER_RATE);

	// Start choosing parents and fill the remaining.
	if (threadIdx.x >= num_parents) {
		// Crossover.
		Chromosome offspring = crossover(blockPopulation, randomState, num_parents);
		// Mutation.
		if (MUTATION_FACTOR > hiprand_uniform(randomState)) {
			mutate(&offspring, randomState);
		}
		// Evaluation.
		offspring.fitnessValue = getFitnessValue(offspring.genes);
		// Updation in population.
		blockPopulation[threadIdx.x] = offspring;
	}
}

/**
 * Core genetic algorithm.
 */
__global__ void geneticAlgorithm(bool freshRun, Chromosome *d_inputPopulation,
		hiprandState* states, Chromosome *d_outputPopulation) {

	__shared__ Chromosome blockPopulation[THREADS_PER_BLOCK];

	int blockIndex = blockIdx.x;
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

	hiprandState randomState = states[threadIndex];

	if (freshRun) {
		// Because this is a stage 1 run, we need to initialize the random population on GPU.
		initializeBlockPopulation(blockPopulation, states);
	} else {
		// TODO: Handle the else case. Might need to copy from global to shared.
	}

	// Barrier ensures that population is available on the block in whatever way.
	__syncthreads();

	for (int z = 0; z < NUM_EPOCHS; z++) {

		if (threadIdx.x == 0) {
			bubbleSort(blockPopulation);
		}
		__syncthreads();
		// Chromosomes orted in the increasing order of fitness function.

		startIteration(blockPopulation, &randomState);
		__syncthreads();
	}

	// all threads of this block have completed.
	__syncthreads();

	if (threadIdx.x == 0) {
		printBlockPopulation(blockPopulation);
		printf("Epochs have been completed. Here's the block's best output:\n");
		bubbleSort(blockPopulation);
		printChromosome(blockPopulation[0]);

		// TODO: Copy these results to the global memory.
	}
	__syncthreads();
}

int main() {
	int NUM_TOTAL_THREADS = NUM_BLOCKS * THREADS_PER_BLOCK;

	hiprandState *d_randomStates = NULL;
	hipMalloc((void**) &d_randomStates,
			NUM_TOTAL_THREADS * sizeof(hiprandState));
	setupRandomStream<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(time(NULL),
			d_randomStates);
	hipDeviceSynchronize();
	printf("CudaStatus: %s\n", hipGetErrorString(hipGetLastError()));

	geneticAlgorithm<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(true, NULL,
			d_randomStates, NULL);

// Freeing the resources.
	hipDeviceSynchronize();
	printf("CudaStatus: %s\n", hipGetErrorString(hipGetLastError()));
	hipFree(d_randomStates);

	return 0;
}
