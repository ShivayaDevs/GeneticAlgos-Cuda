#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <time.h>

#define THREADS_PER_BLOCK 100
#define NUM_BLOCKS 1

using namespace std;
typedef double HighlyPrecise;

/**
 * TODO: Don't know why this does not give good results on gene length >= 15.
 * Most probably random numbers error.
 */
const int GENOME_LENGTH = 14;

const float MUTATION_FACTOR = 0.2;
const float CROSSOVER_RATE = 0.6;

const int NUM_EPOCHS = 5000;

struct Chromosome {
	HighlyPrecise genes[GENOME_LENGTH];
	HighlyPrecise fitnessValue;
};

__global__ void setupRandomStream(unsigned int seed, hiprandState* states) {
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	/* Make sure that this is seed and might want to reduce the number of states to threadIdx.x .*/
	hiprand_init(seed, threadIndex, 0, &states[threadIndex]);
}

__device__ HighlyPrecise getFitnessValue(HighlyPrecise chromosome[]) {
	HighlyPrecise fitnessValue = 0;
	for (int i = 0; i < GENOME_LENGTH; i++) {
		fitnessValue += chromosome[i] * chromosome[i];
	}
	return fitnessValue;
}

__device__ void bubbleSort(Chromosome a[]) {
	int n = THREADS_PER_BLOCK;
	Chromosome temp;
	for (int i = 0; i < n; i++) {
		bool changed = false;
		for (int j = 0; j < n - 1 - i; j++) {
			if (a[j].fitnessValue > a[j + 1].fitnessValue) {
				temp = a[j + 1];
				a[j + 1] = a[j];
				a[j] = temp;
				changed = true;
			}
		}
		if (!changed) {
			// ADD OPTIMIZATION.
			// break;
		}
	}
}

__device__ void printBlockPopulation(Chromosome blockPopulation[]) {
	for (int i = 0; i < THREADS_PER_BLOCK; i++) {
		printf("Fitness: %lf | Chromosome: ", blockPopulation[i].fitnessValue);
		for (int j = 0; j < GENOME_LENGTH; j++) {
			printf("%.02lf ,", blockPopulation[i].genes[j]);
		}
		printf("\n");
	}
}

__global__ void geneticAlgorithm(hiprandState* states) {
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState randomState = states[threadIndex];

	__shared__ Chromosome blockPopulation[THREADS_PER_BLOCK];

	HighlyPrecise chromosome[GENOME_LENGTH];

	for (int i = 0; i < GENOME_LENGTH; i++) {
		chromosome[i] = 2.0 * hiprand_uniform(&randomState) - 1;
		blockPopulation[threadIdx.x].genes[i] = chromosome[i];
	}
	blockPopulation[threadIdx.x].fitnessValue = getFitnessValue(chromosome);

	__syncthreads();

	for (int z = 0; z < NUM_EPOCHS; z++) {

		if ((threadIdx.x == 0) == 1) {
//			printf("==> Before sorting:\n");
//			printBlockPopulation(blockPopulation);
			bubbleSort(blockPopulation);
//			printf("  ==> After sorting:\n");
//			printBlockPopulation(blockPopulation);
		}

		__syncthreads();

		int num_parents = THREADS_PER_BLOCK * (1 - CROSSOVER_RATE);
		if (threadIdx.x >= num_parents) {
//			printf("threadIdx = %d\n", threadIdx.x);
			int maleIndex = hiprand_uniform(&randomState) * num_parents;
			int femaleIndex = hiprand_uniform(&randomState) * num_parents;

//			printf("Inside, \tmale:%d = %d \tfemale:%d = %d\n", maleIndex, maleIndex%num_parents, femaleIndex, femaleIndex % num_parents);

			if (maleIndex == femaleIndex) {
				continue;
			}

			Chromosome male = blockPopulation[maleIndex];
			Chromosome female = blockPopulation[femaleIndex];
			Chromosome offspring;
			for (int i = 0; i < GENOME_LENGTH; i++) {
				offspring.genes[i] =
						(i < GENOME_LENGTH / 2) ?
								male.genes[i] : female.genes[i];
			}

			HighlyPrecise random0To1 = hiprand_uniform(&randomState);
			if (MUTATION_FACTOR > random0To1) {
				for (int i = 0; i < GENOME_LENGTH; i++) {
					HighlyPrecise multiplier = (2.0
							* hiprand_uniform(&randomState) - 1) / 10;
					if (multiplier < -0.1 || multiplier > 0.1) {
						printf("Invalid multiplier: %lf", multiplier);
					}
					offspring.genes[i] *= multiplier;
				}
			}
			offspring.fitnessValue = getFitnessValue(offspring.genes);
//			if (offspring.fitnessValue == male.fitnessValue
//					|| offspring.fitnessValue == female.fitnessValue) {
//				printf("Baccha on maa baap %lf\n", offspring.fitnessValue);
//			} else {
//				printf("NOT on\n");
//			}
			blockPopulation[threadIdx.x] = offspring;
		}
		__syncthreads();
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		printBlockPopulation(blockPopulation);
		printf("Epochs have been completed. Here's the block's best output:");
		bubbleSort(blockPopulation);
		for (int j = 0; j < GENOME_LENGTH; j++) {
			printf("%lf ", blockPopulation[0].genes[j]);
		}
		printf("\nFitness:%e\n", blockPopulation[0].fitnessValue);
	}
	__syncthreads();
}

int main() {
	int NUM_TOTAL_THREADS = NUM_BLOCKS * THREADS_PER_BLOCK;

	hiprandState *d_randomStates = NULL;
	hipMalloc((void**) &d_randomStates,
			NUM_TOTAL_THREADS * sizeof(hiprandState));
	setupRandomStream<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(time(NULL),
			d_randomStates);
	hipDeviceSynchronize();
	printf("%s", hipGetErrorString(hipGetLastError()));

	geneticAlgorithm<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_randomStates);

// Freeing the resources.
	hipDeviceSynchronize();
	printf("%s", hipGetErrorString(hipGetLastError()));
	hipFree(d_randomStates);

	return 0;
}
