#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <time.h>
using namespace std;

#define THREADS_PER_BLOCK 32
#define NUM_BLOCKS 32

typedef double HighlyPrecise;

const int GENOME_LENGTH = 14;
const int GENE_MAX = 1;

const float MUTATION_FACTOR = 0.2;
const float CROSSOVER_RATE = 0.6;

const int NUM_EPOCHS = 1000;

struct Chromosome {
	HighlyPrecise genes[GENOME_LENGTH];
	HighlyPrecise fitnessValue;
};

__global__ void setupRandomStream(unsigned int seed, hiprandState* states) {
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(seed, threadIndex, 0, &states[threadIndex]);
}

__device__ HighlyPrecise getFitnessValue(HighlyPrecise chromosome[]) {
	HighlyPrecise fitnessValue = 0;
	for (int i = 0; i < GENOME_LENGTH; i++) {
		fitnessValue += chromosome[i] * chromosome[i];
	}
	return fitnessValue;
}

/**
 * Sorts the population that is present in the shared memory of one block.
 * Please note that this sorting is sequential.
 */
__device__ void bubbleSort(Chromosome a[]) {
	int n = blockDim.x;
	Chromosome temp;
	for (int i = 0; i < n; i++) {
		bool changed = false;
		for (int j = 0; j < n - 1 - i; j++) {
			if (a[j].fitnessValue > a[j + 1].fitnessValue) {
				temp = a[j + 1];
				a[j + 1] = a[j];
				a[j] = temp;
				changed = true;
			}
		}
		if (!changed) {
			break;
		}
	}
}

__device__ void printChromosome(Chromosome c) {
	printf("Fitness: %lf | Chromosome: ", c.fitnessValue);
	for (int j = 0; j < GENOME_LENGTH; j++) {
		printf("%lf ,", c.genes[j]);
	}
	printf("\n");
}

/**
 * Prints the whole population of a block from the shared memory.
 */
__device__ void printBlockPopulation(Chromosome blockPopulation[]) {
	for (int i = 0; i < blockDim.x; i++) {
		printChromosome(blockPopulation[i]);
	}
}

__device__ void initializeBlockPopulation(Chromosome blockPopulation[],
		hiprandState* randomState) {
	HighlyPrecise chromosome[GENOME_LENGTH];
	for (int i = 0; i < GENOME_LENGTH; i++) {
		chromosome[i] = GENE_MAX * (2.0 * hiprand_uniform(randomState) - 1);
		blockPopulation[threadIdx.x].genes[i] = chromosome[i];
	}
	blockPopulation[threadIdx.x].fitnessValue = getFitnessValue(chromosome);
}

__device__ Chromosome crossover(Chromosome blockPopulation[],
		hiprandState* randomState, int num_parents) {
	// Choosing parents.
	int maleIndex = hiprand_uniform(randomState) * num_parents;
	int femaleIndex = hiprand_uniform(randomState) * num_parents;
	if (maleIndex == femaleIndex) {
		return blockPopulation[threadIdx.x];
	}
	Chromosome male = blockPopulation[maleIndex];
	Chromosome female = blockPopulation[femaleIndex];
	Chromosome offspring;

	for (int i = 0; i < GENOME_LENGTH; i++) {
		offspring.genes[i] =
				(i < GENOME_LENGTH / 2) ? male.genes[i] : female.genes[i];
	}
	return offspring;
}

__device__ void mutate(Chromosome *offspring, hiprandState* randomState) {
	for (int i = 0; i < GENOME_LENGTH; i++) {
		HighlyPrecise multiplier = (2.0 * hiprand_uniform(randomState) - 1) / 10;
		offspring->genes[i] *= multiplier;
	}
}

__device__ void startIteration(Chromosome blockPopulation[],
		hiprandState* randomState) {

	int num_parents = blockDim.x * (1 - CROSSOVER_RATE);

	// Start choosing parents and fill the remaining.
	if (threadIdx.x >= num_parents) {
		// Crossover.
		Chromosome offspring = crossover(blockPopulation, randomState,
				num_parents);
		// Mutation.
		if (MUTATION_FACTOR > hiprand_uniform(randomState)) {
			mutate(&offspring, randomState);
		}
		// Evaluation.
		offspring.fitnessValue = getFitnessValue(offspring.genes);
		// Updation in population.
		blockPopulation[threadIdx.x] = offspring;
	}
}

/**
 * Core genetic algorithm.
 */
__global__ void geneticAlgorithm(bool freshRun, Chromosome *d_inputPopulation,
		hiprandState* states, Chromosome *d_outputPopulation) {

	/* This won't be the same size when running in stage 2,
	 * i.e. Block's bests conpetiting against each other.
	 * But since CUDA doesn't allow (in an easy way) to dynamically decide the size,
	 * it has been kept to the larger one. BUT, blocks can be more than threads.*/
	__shared__ Chromosome blockPopulation[THREADS_PER_BLOCK];

	int blockIndex = blockIdx.x;
	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

	hiprandState randomState = states[threadIndex];

	if (freshRun) {
		// Because this is a stage 1 run, we need to initialize the random population on GPU.
		initializeBlockPopulation(blockPopulation, states);
	} else {
		blockPopulation[threadIdx.x] = d_inputPopulation[threadIndex];
	}

	// Barrier ensures that population is available on the block in whatever way.
	__syncthreads();

	for (int z = 0; z < NUM_EPOCHS; z++) {

		if (threadIdx.x == 0) {
			bubbleSort(blockPopulation);
		}
		__syncthreads();
		// Chromosomes orted in the increasing order of fitness function.

		startIteration(blockPopulation, &randomState);
		__syncthreads();
	}

	// all threads of this block have completed.
	__syncthreads();

	if (threadIdx.x == 0) {
		bubbleSort(blockPopulation);

		// Copy these results to the global memory.
		d_outputPopulation[blockIndex] = blockPopulation[0];
	}
	__syncthreads();
}

void checkForCudaErrors() {
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("Warning: %s\n", hipGetErrorString(hipGetLastError()));
	}
}

int main() {
	int NUM_TOTAL_THREADS = NUM_BLOCKS * THREADS_PER_BLOCK;

	// Setup the random number generation stream on device.
	hiprandState *d_randomStates = NULL;
	hipMalloc((void**) &d_randomStates,
			NUM_TOTAL_THREADS * sizeof(hiprandState));
	setupRandomStream<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(time(NULL),
			d_randomStates);
	hipDeviceSynchronize();
	checkForCudaErrors();

	Chromosome *h_gpuOut = NULL;
	Chromosome *d_outputPopulation = NULL;
	h_gpuOut = (Chromosome*) malloc(NUM_BLOCKS * sizeof(Chromosome));
	hipMalloc((void**) &d_outputPopulation, NUM_BLOCKS * sizeof(Chromosome));
	hipDeviceSynchronize();
	checkForCudaErrors();

	geneticAlgorithm<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(true, NULL,
			d_randomStates, d_outputPopulation);

	hipMemcpy(h_gpuOut, d_outputPopulation, sizeof(Chromosome) * NUM_BLOCKS,
			hipMemcpyDeviceToHost);

	//	stage2
	Chromosome *d_inputPopulation;
	hipMalloc((void**) &d_inputPopulation, sizeof(Chromosome) * NUM_BLOCKS);
	hipDeviceSynchronize();
	checkForCudaErrors();

	hipMemcpy(d_inputPopulation, h_gpuOut, sizeof(Chromosome) * NUM_BLOCKS,
			hipMemcpyHostToDevice);

	geneticAlgorithm<<<1, NUM_BLOCKS>>>(false, d_inputPopulation,
			d_randomStates, d_outputPopulation);

	hipMemcpy(h_gpuOut, d_outputPopulation, sizeof(Chromosome) * 1,
			hipMemcpyDeviceToHost);
	printf("======== GPU Results ========\n");
	printf("==> Best Fitness Value: %e\n\nBest Chromosome: ", h_gpuOut[0].fitnessValue);
	for (int i = 0; i < GENOME_LENGTH; i++) {
		printf("%e ", h_gpuOut[0].genes[i]);
	}
	printf("\n");

	hipDeviceSynchronize();
	checkForCudaErrors();

	// Freeing the resources.
	hipFree(d_randomStates);
	hipFree(d_outputPopulation);
	return 0;
}
